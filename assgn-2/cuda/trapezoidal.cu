/*
Title- Numerical integration of sin function using Trapezoidal method (parallel execution)
Author- Harshit Agrawal
Description- This integrates the sin function using Trapezoidal method in [0,PI] parallely using CUDA library.
To compile using openmpi run "$ nvcc trapezoidal.c -o trapezoidal.out"
To run the executable run "$ ./montecarlo.out"
Change the value of N, THREADS_PER_BLOCK according to the experimental setup.
*/



#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<time.h>
#define N 1000000
#define THREADS_PER_BLOCK (100)
#define DX M_PI/N


//Device function to compute the area of differential element using trapezoidal approximation
__global__ void area( float *dev_sum) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    dev_sum[index] = ((sinpif(((float)index)/N)+sinpif(((float)(index+1))/N))/2)*DX;
}



int main( void ) {
    float *sum, *dev_sum;
    double size = N * sizeof( float );
    hipMalloc( (void**)&dev_sum, size );
    sum = (float*)malloc( size );
    clock_t begin = clock();
    area<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_sum);
    
    hipMemcpy( sum, dev_sum, size, hipMemcpyDeviceToHost );
    float net_sum = 0;
    for (int i=0; i<N; i++){
        net_sum += sum[i];
    }
    clock_t end = clock();
    double timex = (double)(end-begin)/CLOCKS_PER_SEC;
    printf("Total time of execution: %f \n", timex);
    printf("sum= %f \n", net_sum);
    free( sum );
    hipFree( dev_sum );
    return 0;
}
