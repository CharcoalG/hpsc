/*
Title- Numerical integration of sin function using Monte Carlo method (parallel execution)
Author- Harshit Agrawal
Description- This integrates the sin function using Monte Carlo method in [0,PI] parallely using CUDA library.
To compile using CUDA run "$ nvcc montecarlo.cu -o montecarlo.out"
To run the executable run "$ ./montecarlo.out"
Change the value of N, THREADS_PER_BLOCK according to the experimental setup.
*/

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<time.h>
#define N 1000000
#define THREADS_PER_BLOCK (100)


//This function generates the random numbers
__device__ float generate( hiprandState* globalState, int ind )
{
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

//This function sets up the random number generator
__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init ( seed, id, 0, &state[id] );        
}

//This function executes the main monte carlo method
__global__ void area( int *dev_sum, hiprandState* globalState) {
    __shared__ int success[THREADS_PER_BLOCK];
    float r1 = generate(globalState, threadIdx.x + 2 * blockIdx.x * blockDim.x)*M_PI, r2 = generate(globalState, threadIdx.x + 2*blockIdx.x * blockDim.x+blockDim.x);
    float sinr1 = sinf(r1);
    if (sinr1>=r2){
        success[threadIdx.x] = 1;
    } else{
        success[threadIdx.x] = 0;
    }
    __syncthreads();
    int suc_num = 0;
    if (0==threadIdx.x){
        for (int i=0; i<THREADS_PER_BLOCK; i++){
            if (success[i]){
                ++suc_num;
            }
        }
        dev_sum[blockIdx.x] = suc_num;
    }
}    



int main( void ) {
    int *sum, *dev_sum;
    float integral = 0;
    hiprandState* devStates;
    hipMalloc ( &devStates, N*sizeof( hiprandState ) );
    double size = (N/THREADS_PER_BLOCK)*sizeof(int);
    clock_t begin = clock();

    setup_kernel <<< 2*N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>> ( devStates,unsigned(time(NULL)) );

    hipMalloc( (void**)&dev_sum, size );
    sum = (int*)malloc( size );

    area<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_sum, devStates);

    hipMemcpy( sum, dev_sum, size, hipMemcpyDeviceToHost );
    float net_sum = 0;
    for (int i=0; i<(N/THREADS_PER_BLOCK); i++){
        net_sum += sum[i];
    }

    integral = ((float)net_sum)*M_PI/N;
    clock_t end = clock();
    double timex = (double)(end-begin)/CLOCKS_PER_SEC;
    printf("Total time of execution: %f \n", timex);
    printf("integral= %f \n", integral);
    free( sum );
    hipFree( dev_sum ), hipFree(devStates);
    return 0;
}
